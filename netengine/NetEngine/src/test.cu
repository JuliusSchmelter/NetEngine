#include <algorithm>
#include <iostream>
#include <math.h>
#include <thread>

#include "Exceptions.h"
#include "Net.h"

//--------------------------------------------------------------------------------------------------
// Test accuracy.
//--------------------------------------------------------------------------------------------------
float NetEngine::Net::test(const std::vector<std::vector<float>>& samples,
                           const std::vector<std::vector<uint32_t>>& labels) {
    // Check for bad inputs.
    if (samples.size() != labels.size())
        throw NetEngine::SetSizeError(samples.size(), labels.size());

    // Allocate device memory for (intermediate) results.
    float* results_d[m_weights.size()];
    for (size_t i = 0; i < m_weights.size(); i++) {
        TRY_CUDA(hipMalloc(&results_d[i], m_weights[i].rows * sizeof(float)));
    }

    // Allocate device memory for sample.
    float* sample_d;
    TRY_CUDA(hipMalloc(&sample_d, samples[0].size() * sizeof(float)));

    // Allocate host memory for output.
    std::vector<float> output(m_layout.back());

    // Iterate samples.
    size_t success = 0;
    for (size_t i = 0; i < samples.size(); i++) {
        // Copy sample to device memory.
        TRY_CUDA(hipMemcpy(sample_d, samples[i].data(), samples[i].size() * sizeof(float),
                            hipMemcpyHostToDevice));

        // Run sample on network.
        run_cuda(sample_d, results_d);

        // Copy output to host.
        TRY_CUDA(hipMemcpy(output.data(), results_d[m_weights.size() - 1],
                            m_layout.back() * sizeof(float), hipMemcpyDeviceToHost));

        // Check result.
        uint8_t result = std::max_element(output.begin(), output.end()) - output.begin();
        uint8_t label = std::max_element(labels[i].begin(), labels[i].end()) - labels[i].begin();

        if (result == label)
            success++;
    }

    // Free memory.
    TRY_CUDA(hipFree(sample_d));
    for (size_t i = 0; i < m_weights.size(); i++) {
        TRY_CUDA(hipFree(results_d[i]));
    }

    return (float)success / (float)samples.size();
}
